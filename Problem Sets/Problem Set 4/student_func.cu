#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "reference_calc.cpp"
#include "utils.h"

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__ void histogram(unsigned int* const d_inputVals,
                          unsigned int* d_bins,
                          unsigned int* const d_cdf,
                          int bit,
                          const size_t numElems)
{
 int index = blockDim.x * blockIdx.x + threadIdx.x;
 int tid = threadIdx.x;
 if(index > int(numElems)) return;
 
 __shared__ unsigned int s_bins[2];
 
 if(tid == 0)
 {
  s_bins[0] = 0;
  s_bins[1] = 0;
 }
 __syncthreads();
 int bin = (d_inputVals[index] >> bit) & 1;
 atomicAdd(&s_bins[bin],1);
 __syncthreads();
 
 if(tid == 0)
 {
 atomicAdd(&d_bins[0] , s_bins[0]);
 atomicAdd(&d_bins[1] , s_bins[1]);
 atomicAdd(&d_cdf[1], s_bins[0]);
 }
 
 
 
}

__global__ void exclusiveSum(unsigned int* const d_cdf, 
                             unsigned int* d_bins)
{
 int acc = 0;

 for(int i = 0; i < 2; i++)
 {
  d_cdf[i] = acc;
  acc = acc + d_bins[i];
 }
}

/*
__global__ void offset(unsigned int* const d_inputVals,
                       int bit,
                       int* d_offset,
                       const size_t numElems)
{
 int counter0 = 0;
 int counter1 = 0;
 for(int i = 0; i < int(numElems); i++)
 {
  int bin = (d_inputVals[i]>>bit)&1;
  if(bin == 0)
  {
   d_offset[i] = counter0;
   counter0++;
  }
  if(bin == 1)
  {
    d_offset[i] = counter1;
    counter1++;
  }
 }
}
*/




__global__ void scanIn(unsigned int * const d_inputVals,
                       int bit,
                       int * d_scanInOnes,
                       int * d_scanInZeroes,
                       const size_t numElems)
{
 int index = threadIdx.x + blockDim.x * blockIdx.x;
 
 if(index > numElems) return;
 
 if(((d_inputVals[index]>>bit)&1) == 0) d_scanInZeroes[index] = 1;
 else d_scanInOnes[index] = 1;
}

/*
__global__ void scanOut(unsigned int * const d_inputVals,
                       int bit,
                       int * d_scanOutOnes,
                       int * d_scanOutZeroes,
                       int * d_scanInOnes,
                       int * d_scanInZeroes,
                       const size_t numElems)
{
 int acc0 = 0;
 int out0 = 0;
 int acc1 = 0;
 int out1 = 0;
 int index = threadIdx.x + blockDim.x * blockIdx.x;
 int tid = threadIdx.x;
 
 if(index > numElems) return;
 
 extern __shared__ int s_scanIn[];
 
 if(tid == 0)
 {
  for(int i = 0; i < numElems; i++)
  {
    s_scanIn[i] = d_scanInZeroes[i];
    //s_scanIn[i+numElems] = d_scanInOnes[i];
  }
 }
 __syncthreads();
 
 if(((d_inputVals[index]>>bit)&1) == 0)
 {
  for(int i = 0; i < index; i++)
  {
   acc0 = acc0 + s_scanIn[i];
   out0 = acc0;
  }
  d_scanOutZeroes[index] = out0;
 }
 else 
 {
  for(int i = 0; i < index; i++)
  {
   acc1 = acc1 + s_scanIn[i+numElems];
   out1 = acc1;
  }
  d_scanOutOnes[index] = out1;
 }
}
*/
__global__ void scanOut(int * d_scanOut,
                        int * d_scanIn,
                        const size_t numElems,
                        int jump)
{
 int index = threadIdx.x + blockDim.x * blockIdx.x;
 
 if(index > numElems) return;
 
 if(index>=jump) d_scanOut[index] = d_scanIn[index] + d_scanIn[index-jump];
 else d_scanOut[index] = d_scanIn[index];
 __syncthreads();
}


__global__ void cpy(int * d_scanOut,
                    int * d_scanIn,
                    const size_t numElems)
{
 int index = blockDim.x * blockIdx.x + threadIdx.x;
 if(index > int(numElems)) return;
 
  d_scanIn[index] = d_scanOut[index];
 
}

__global__ void offset(int bit, int * d_offset,
                       int * d_scanOutOnes,
                       int * d_scanOutZeroes,
                       int * d_scanInOnes,
                       int * d_scanInZeroes,
                       const size_t numElems)
{
 int index = threadIdx.x + blockDim.x * blockIdx.x;
 
 if(index > numElems) return;
 
 if(d_scanInOnes[index] == 1)
 {
  d_offset[index] = d_scanOutOnes[index]-1;
 }
 else 
 {
  d_offset[index] = d_scanOutZeroes[index]-1;
 }
}

__global__ void radixSort(unsigned int* const d_inputVals,
                          unsigned int* const d_inputPos,
                          unsigned int* const d_outputVals,
                          unsigned int* const d_outputPos,
                          unsigned int* const d_cdf,
                          int* d_offset,
                          int bit,
                          const size_t numElems)
{
 int index = blockDim.x * blockIdx.x + threadIdx.x;
 
 if(index > int(numElems)) return;
 
  int bin = (d_inputVals[index] >> bit) & 1;
  int idx = d_offset[index] + d_cdf[bin];
  
  d_outputVals[idx] = d_inputVals[index];
  d_outputPos[idx] = d_inputPos[index];
 
}

__global__ void cpyArray(unsigned int* const d_inputVals,
                         unsigned int* const d_inputPos,
                         unsigned int* const d_outputVals,
                         unsigned int* const d_outputPos,
                         const size_t numElems)
{
 int index = blockDim.x * blockIdx.x + threadIdx.x;
 if(index > int(numElems)) return;
 
  d_inputVals[index] = d_outputVals[index];
  d_inputPos[index] = d_outputPos[index];
 
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
 unsigned int h_bins[2];
 unsigned int h_cdf[2];
 int h_offset[numElems];
 int h_scanInOnes[numElems];
 int h_scanInZeroes[numElems];
 int h_scanOutOnes[numElems];
 int h_scanOutZeroes[numElems];
 
 for(int i = 0; i < 2; i++)
 {
  h_bins[i] = 0;
  h_cdf[i] = 0;
 }
 
 for(int i = 0; i < int(numElems); i++)
 {
 h_offset[i] = 0;
 h_scanInOnes[i] = 0;
 h_scanInZeroes[i] = 0;
 h_scanOutOnes[i] = 0;
 h_scanOutZeroes[i] = 0;
 }
 

 unsigned int* d_bins;
 unsigned int* d_cdf;
 //unsigned int* d_bit;
 int* d_offset;
 int* d_scanInOnes;
 int* d_scanInZeroes;
 int* d_scanOutOnes;
 int* d_scanOutZeroes;
 int* d_ones;
 int* d_zeroes;

 checkCudaErrors(hipMalloc((void**) &d_bins, 2 * sizeof(unsigned int)));
 checkCudaErrors(hipMemcpy(d_bins, h_bins, 2 * sizeof(unsigned int), hipMemcpyHostToDevice));
 
 checkCudaErrors(hipMalloc((void**) &d_cdf, 2 * sizeof(unsigned int)));
 checkCudaErrors(hipMemcpy(d_cdf, h_cdf, 2 * sizeof(unsigned int), hipMemcpyHostToDevice));
 
 checkCudaErrors(hipMalloc((void**) &d_offset, numElems * sizeof(int)));
 checkCudaErrors(hipMemcpy(d_offset, h_offset, numElems * sizeof(int), hipMemcpyHostToDevice));
 
 checkCudaErrors(hipMalloc((void**) &d_scanInOnes, numElems * sizeof(int)));
 checkCudaErrors(hipMemcpy(d_scanInOnes, h_scanInOnes, numElems * sizeof(int), hipMemcpyHostToDevice));
 
 checkCudaErrors(hipMalloc((void**) &d_scanInZeroes, numElems * sizeof(int)));
 checkCudaErrors(hipMemcpy(d_scanInZeroes, h_scanInZeroes, numElems * sizeof(int), hipMemcpyHostToDevice));
 
 checkCudaErrors(hipMalloc((void**) &d_scanOutOnes, numElems * sizeof(int)));
 checkCudaErrors(hipMemcpy(d_scanOutOnes, h_scanOutOnes, numElems * sizeof(int), hipMemcpyHostToDevice));
 
 checkCudaErrors(hipMalloc((void**) &d_scanOutZeroes, numElems * sizeof(int)));
 checkCudaErrors(hipMemcpy(d_scanOutZeroes, h_scanOutZeroes, numElems * sizeof(int), hipMemcpyHostToDevice));
 
 checkCudaErrors(hipMalloc((void**) &d_zeroes, numElems * sizeof(int)));
 checkCudaErrors(hipMalloc((void**) &d_ones, numElems * sizeof(int)));

 
 //checkCudaErrors(hipMalloc((void**) &d_bit, sizeof(unsigned int)));

 int blockLength = 32;
 const dim3 blockSize(blockLength, 1, 1);
 const dim3 gridSize(ceil(numElems/blockLength), 1, 1);
 
 for (unsigned int bit = 0; bit < 32; bit++)
 {
 
  checkCudaErrors(hipMemset(d_cdf, 0,  2 * sizeof(unsigned int)));
  checkCudaErrors(hipMemset(d_bins, 0,  2 * sizeof(unsigned int)));
  checkCudaErrors(hipMemset(d_offset, 0,  numElems * sizeof(int)));
  checkCudaErrors(hipMemset(d_scanInOnes, 0,  numElems * sizeof(int)));
  checkCudaErrors(hipMemset(d_scanInZeroes, 0,  numElems * sizeof(int)));
  checkCudaErrors(hipMemset(d_scanOutOnes, 0,  numElems * sizeof(int)));
  checkCudaErrors(hipMemset(d_scanOutZeroes, 0,  numElems * sizeof(int)));
  
  //checkCudaErrors(hipMemcpy(d_bit, &bit, sizeof(unsigned int), hipMemcpyHostToDevice));
 
  histogram<<<gridSize, blockSize>>>(d_inputVals, d_bins, d_cdf, bit, numElems);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  //exclusiveSum<<<1, 1>>>(d_cdf, d_bins);
  //hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  scanIn<<<gridSize, blockSize>>>(d_inputVals, bit, d_scanInOnes, d_scanInZeroes, numElems);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  hipMemcpy(d_ones, d_scanInOnes, numElems * sizeof(int), hipMemcpyDeviceToDevice);
  hipMemcpy(d_zeroes, d_scanInZeroes, numElems * sizeof(int), hipMemcpyDeviceToDevice);
  
  for(int jump = 1; jump < numElems; jump<<=1)
  {
   scanOut<<<gridSize, blockSize>>>(d_scanOutZeroes, d_zeroes,
                                    numElems, jump);
   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
   
   cpy<<<gridSize, blockSize>>>(d_scanOutZeroes, d_zeroes, numElems);
   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  }
  
  for(int jump = 1; jump < numElems; jump<<=1)
  {
   scanOut<<<gridSize, blockSize>>>(d_scanOutOnes, d_ones,
                                    numElems, jump);
   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
   
   cpy<<<gridSize, blockSize>>>(d_scanOutOnes, d_ones, numElems);
   hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  }
  
  offset<<<gridSize, blockSize>>>(bit, d_offset, d_scanOutOnes, d_scanOutZeroes,
                                  d_scanInOnes, d_scanInZeroes, numElems);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
 
  radixSort<<<gridSize, blockSize>>>(d_inputVals, d_inputPos,
                                     d_outputVals, d_outputPos,
                                     d_cdf, d_offset, bit, numElems);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  cpyArray<<<gridSize, blockSize>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
 }
 
 hipFree(d_bins);
 hipFree(d_cdf);
 hipFree(d_offset);
 hipFree(d_scanInOnes);
 hipFree(d_scanInZeroes);
 hipFree(d_scanOutOnes);
 hipFree(d_scanOutZeroes);

}
